#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <chrono>
#define NUM_THREADS 128


int *number_sum_h;
int *number_sum_d;
int *numbers_d;
// int *numbers_h;
int blks;

__global__ void generate_numbers(int* numbers_d, int size){
     int tid = threadIdx.x + blockIdx.x * blockDim.x;
     if (tid >= size){
        return;
     }
    numbers_d[tid] = tid % 10;
    // printf("%d \n",tid);
}
__global__ void stencil(int* numbers_d, int size, int *number_sum_d){
     int tid = threadIdx.x + blockIdx.x * blockDim.x;
     if (tid >= size){
        return;
     }
     int num0=0;
     int num1=0;
     int num2=0;
     int num3=0;
     int num4=0;
     int num5=0;
     if(tid > 1){
        num0 = numbers_d[tid-2];
     }
     if(tid > 0){
        num1 = numbers_d[tid-1];
     }
     if(tid < size -2){
        num5 = numbers_d[tid+2];
     }
    if(tid < size -1){
        num4 = numbers_d[tid+1];
     }
    num3 = numbers_d[tid];
    number_sum_d[tid] = num0 + num1 + num2 + num3 + num4 + num5;
}

int find_arg_idx(int argc, char** argv, const char* option) {
    for (int i = 1; i < argc; ++i) {
        if (strcmp(argv[i], option) == 0) {
            return i;
        }
    }
    return -1;
}

int find_int_arg(int argc, char** argv, const char* option, int default_value) {
    int iplace = find_arg_idx(argc, argv, option);

    if (iplace >= 0 && iplace < argc - 1) {
        return std::stoi(argv[iplace + 1]);
    }

    return default_value;
}


int main(int argc, char** argv){
    int N = find_int_arg(argc, argv, "-n", 1000);
    printf("%d \n", N);
    float quotient;
    quotient = float(N/NUM_THREADS);
    blks = ceil(int(quotient)) + 1;
    hipMalloc(&numbers_d, N * sizeof(int));
    hipMalloc(&number_sum_d, N * sizeof(int));
    generate_numbers <<<blks ,NUM_THREADS>>>(numbers_d, N);
    hipDeviceSynchronize();
    auto start_time = std::chrono::steady_clock::now();
    stencil<<<blks,NUM_THREADS>>>(numbers_d,N, number_sum_d);
    hipDeviceSynchronize();
    auto end_time = std::chrono::steady_clock::now();
    std::chrono::duration<double> diff = end_time - start_time;
    double seconds = diff.count();
    std::cout << "Simulation Time = " << seconds << " seconds \n";
    // cudaError_t cudaerr = cudaDeviceSynchronize();
    // if (cudaerr != cudaSuccess)
    //     printf("kernel launch failed with error \"%s\".\n",
    //     cudaGetErrorString(cudaerr));
    number_sum_h = new int[N];
    hipMemcpy(number_sum_h, number_sum_d, N * sizeof(int), hipMemcpyDeviceToHost);

    delete[] number_sum_h;
    hipFree(numbers_d);
    return 0;
}